#include <iostream>
#include <hip/hip_runtime.h>

// CUDA 核函数用于向量加法
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A = new float[numElements];
    float *h_B = new float[numElements];
    float *h_C = new float[numElements];

    // 初始化输入向量
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // 分配 GPU 内存
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将输入数据从主机复制到 GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 启动向量加法核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // 将结果从 GPU 复制回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 验证结果
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            std::cerr << "fail" << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "success" << std::endl;

    // 释放设备和主机内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
