#include "hip/hip_runtime.h"

#include <cstdio>

#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

__device__ double myatomicAdd(double *address, double val)
{
    unsigned long long *address_as_ull = (unsigned long long *)address;
    unsigned long long old = *address_as_ull;
    unsigned long long assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void sumArrayWithAtomic(real *array, real *sum, int N)
{
    // 获取线程的全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // 确保索引在数组范围内
    if (index < N)
    {
        // 使用atomicAdd原子地更新总和  目前是支持double类型 老一些的硬件设备可能不支持 需要利用CAS
        myatomicAdd(sum, array[index]);
    }
}

int main()
{
    const int N = 100; // 数组大小
    const size_t size = N * sizeof(real);

    // 分配和初始化设备内存
    real *d_array, *d_sum;
    hipMalloc(&d_array, size);
    hipMalloc(&d_sum, sizeof(real));
    real h_array[N], h_sum = 0;
    for (int i = 1; i <= N; i++)
    {
        h_array[i - 1] = i;
    }
    // 假设h_array已经初始化
    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(real), hipMemcpyHostToDevice);

    // 启动kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    sumArrayWithAtomic<<<blocksPerGrid, threadsPerBlock>>>(d_array, d_sum, N);

    // 将结果复制回主机
    hipMemcpy(&h_sum, d_sum, sizeof(real), hipMemcpyDeviceToHost);

    // 清理
    hipFree(d_array);
    hipFree(d_sum);

    printf("Total sum: %f\n", h_sum);

    return 0;
}
